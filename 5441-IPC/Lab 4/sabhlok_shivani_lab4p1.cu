#include "hip/hip_runtime.h"
/*
This is a cuda parallel vesion program to multiply a matrix with its transpose on the GPU
This will also multiply on the CPU to verify the reults
The matrix will be generated internally with numbers between [1.0,2.0]
*/
#include<time.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>
#define THRESHOLD .00001

//thread hierarchy
int blocks[] = {2,4,8,16,16};
int threadsPerBlock[] = {1024,512,1024,512,1024};

//matrix multiplied with its transpose on GPU device.
__global__ void MatrixTransposeMultiply_Device(double *A, double *C, int dim)
{
	for (int i = blockIdx.x; i < dim; i += gridDim.x) {
		for (int j = threadIdx.x; j < dim; j += blockDim.x) {
        double sum = 0.0;

        for (int k = 0; k < dim; k++) {
            double a = A[k * dim + i];
            double b = A[k * dim + j];
            sum += a * b;
        }
        C[i * dim + j] = sum;
    }
	}
}

//matrix multiplied with its transpose on CPU.
void MatrixTransposeMultiply_Host(double A[][1024], double C[][1024], int dim) {
    for (int i = 0; i < dim; i++)
        for (int j = 0; j < dim; j++)
        {
            double sum=0;
            for(int k = 0; k < dim; k++)
                sum += A[k][i] * A[k][j];
            C[i][j] = sum;
        }
}

//verification on CPU
int MatrixTransposeMultiply_HostValidate(double A[][1024], double C[][1024], int dim)
{
    int goodResult = 1;
    for (int i = 0; i < dim; i++)
        for (int j = 0; j < dim; j++) {
            if( abs(C[i][j] - A[i][j]) > THRESHOLD)
            {
                goodResult = 0;
                break;
            }
        }
    return goodResult;
}

int main(void) {
    double A[1024][1024], C[1024][1024],result[1024][1024];
    int dim = 1024;
    double *d_A, *d_C;
    size_t memSize = dim * dim * sizeof(double);

    //initializing matrix with random values between 1.0 to 2.0
    for (int i= 0; i< dim; i++)
        for (int j = 0; j < dim; j++)
            A[i][j] = ((double)rand() / RAND_MAX) + 1.0;

    //Allocate and initialize device memory
    hipMalloc( (void**) &d_A, memSize);
    hipMalloc( (void**) &d_C, memSize);
    hipMemcpy(d_A, A, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, memSize, hipMemcpyHostToDevice);

    //timing constructs 
    struct timeval TIMEVAL;
	struct timezone TIMEZONE;
    gettimeofday (&TIMEVAL, &TIMEZONE);
	double startTime = (TIMEVAL.tv_sec + TIMEVAL.tv_usec*1.0e-6);
    //multiplication on the host
    MatrixTransposeMultiply_Host(A, C, dim);
    gettimeofday (&TIMEVAL, &TIMEZONE);
    double endTime = (TIMEVAL.tv_sec + TIMEVAL.tv_usec*1.0e-6);
    //performance on CPU
	printf("\n\n***********************************************************************\n");
    printf("Time taken on CPU (sec) = %lf \n",endTime- startTime);
    printf("GFLOPS = %lf\n\n", (2L * dim * dim * (dim-1))/ (1e9*(endTime- startTime)));

    //Launch kernel
	for(int i = 0; i< 5;i++)
    {
		dim3 dimGrid(blocks[i]);
		dim3 dimBlock(threadsPerBlock[i]);
		startTime = (TIMEVAL.tv_sec + TIMEVAL.tv_usec*1.0e-6);
		//multiplication on device
		MatrixTransposeMultiply_Device<<<dimGrid, dimBlock>>>(d_A, d_C, dim);
		//cuda synchronize
		hipDeviceSynchronize();
		gettimeofday (&TIMEVAL, &TIMEZONE);
		endTime = (TIMEVAL.tv_sec + TIMEVAL.tv_usec*1.0e-6);
		//performance on GPU
		printf("With %d blocks and %d threads per block, Time taken on GPU (sec) = %lf \n",blocks[i],threadsPerBlock[i], endTime- startTime);
		printf("GFLOPS per sec = %lf\n", (2.0 * dim * dim * (dim-1))/ (1e9*(endTime- startTime)));
	}
    //copy results back to host
    hipMemcpy(result, d_C, memSize, hipMemcpyDeviceToHost);

    //verification
    if(!MatrixTransposeMultiply_HostValidate(C, result, dim))
        fprintf(stderr, "oops\n");
	else
		printf("Results matched \n");
	printf("\n\n***********************************************************************\n");
    //Free memory
    hipFree(d_A);
    hipFree(d_C);
}
