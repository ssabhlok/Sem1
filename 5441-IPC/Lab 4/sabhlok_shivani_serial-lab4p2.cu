#include "hip/hip_runtime.h"
/*
This is a serial vesion program of sobel operator on CPU
*/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "time.h"
#include <math.h>
#include<sys/time.h>
extern "C" {
#include "read_bmp.h"
}

#define MAX_ALLOWED_CMD_ARGS  3

#define MIN_RGB_VALUE 0
#define MAX_RGB_VALUE 255

int main(int argc, char* argv[])
{
    int cmd_arg;
    uint8_t *bmp_data;
    uint8_t *new_bmp_img;
    uint32_t wd, ht;
    uint32_t i, j;
    double sum1, sum2;
	double mag;
    uint32_t threshold;
    FILE *out_file, *inFile;
    uint32_t percent_black_cells = 0;
    uint32_t total_cells;

    /*Check if no of arguments is permissible to execute*/
    if (argc != MAX_ALLOWED_CMD_ARGS)
    {
        fprintf(stderr, "Missing input file name and/or output file name in command line parameters\n");
        exit(EXIT_FAILURE);
    }
    /*Reading command line arguments and obtain the values*/
    for (cmd_arg = 1; cmd_arg < argc; cmd_arg++)
    {
        switch (cmd_arg)
        {
            case 1:
                inFile = fopen(argv[cmd_arg], "rb");
                break;

            case 2:
                out_file = fopen(argv[cmd_arg], "wb");
                break;
        }
    }
    //Read the binary bmp file into
    bmp_data = (uint8_t *)read_bmp_file(inFile);
    //Allocate space for new sobel image
    new_bmp_img = (uint8_t *)malloc(get_num_pixel());
    wd = get_image_width();
    ht = get_image_height();
    threshold   = 0;
    total_cells = wd * ht;
	//timing constructs 
    struct timeval TIMEVAL;
	struct timezone TIMEZONE;
    gettimeofday (&TIMEVAL, &TIMEZONE);
	double startTime = (TIMEVAL.tv_sec + TIMEVAL.tv_usec*1.0e-6);
    //serial code
    while(percent_black_cells < 75)
    {
        percent_black_cells = 0;
        threshold += 1;
        for(i=1; i < (ht-1); i++)
        {
            for(j=1; j < (wd-1); j++)
            {
				//applying sobel stencil
                sum1 = bmp_data[ (i-1)*wd + (j+1) ] - bmp_data[ (i-1)*wd + (j-1) ] \
						+ 2*bmp_data[ (i)*wd + (j+1) ] - 2*bmp_data[ (i)*wd + (j-1) ] \
						+ bmp_data[ (i+1)*wd + (j+1) ] - bmp_data[ (i+1)*wd + (j-1) ];

                sum2 = bmp_data[ (i-1)*wd + (j-1) ] + 2*bmp_data[ (i-1)*wd + (j) ] \
						+ bmp_data[ (i-1)*wd + (j+1) ] - bmp_data[ (i+1)*wd + (j-1) ] \
						- 2*bmp_data[ (i+1)*wd + (j) ] - bmp_data[ (i+1)*wd + (j+1) ];

                mag = sqrt(sum1 * sum1 + sum2 * sum2);
                if(mag > threshold)
                {
                    new_bmp_img[ i*wd + j] = MAX_RGB_VALUE;
                }
                else
                {
                    new_bmp_img[ i*wd + j] = MIN_RGB_VALUE;
                    percent_black_cells++;
                }
            }
        }
        percent_black_cells = (percent_black_cells * 100) / total_cells;
    }

    //end bechmark measurement prior to writing out file
    gettimeofday (&TIMEVAL, &TIMEZONE);
    double endTime = (TIMEVAL.tv_sec + TIMEVAL.tv_usec*1.0e-6);
    //performance on CPU
	printf("\n\n***********************************************************************\n");
    printf("Elapsed time for Sobel Operation on CPU (sec): %f\n\n",endTime- startTime);
    printf("Theshold: %d\n",threshold);
	printf("\n\n***********************************************************************\n");
    //Write the buffer into the bmp file
    write_bmp_file(out_file, new_bmp_img);

    return 0;
}