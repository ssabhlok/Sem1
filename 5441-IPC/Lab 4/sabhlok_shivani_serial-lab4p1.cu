/*
This is a serial program to multiply a matrix with its transpose on the CPU
The matrix will be generated internally with numbers between [1.0,2.0]
*/

#include <hip/hip_runtime.h>
#include<time.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>

// matrix multiplied with its transpose on CPU.
void MatrixTransposeMultiply_Host(double A[][1024], double C[][1024], int dim) {
    for (int i = 0; i < dim; i++)
        for (int j = 0; j < dim; j++)
        {
            double sum=0;
            for(int k = 0; k < dim; k++)
                sum += A[k][i] * A[k][j];
            C[i][j] = sum;
        }
}

int main(void) {
    double A[1024][1024], C[1024][1024];
    int dim = 1024;

    //initializing matrix with random values between 1.0 to 2.0
    for (int i= 0; i< dim; i++)
        for (int j = 0; j < dim; j++)
            A[i][j] = ((double)rand() / RAND_MAX) + 1.0;
   //timing constructs 
    struct timeval TIMEVAL;
	struct timezone TIMEZONE;
    gettimeofday (&TIMEVAL, &TIMEZONE);
	double startTime = (TIMEVAL.tv_sec + TIMEVAL.tv_usec*1.0e-6);
    //multiplication on the host
    MatrixTransposeMultiply_Host(A, C, dim);
	gettimeofday (&TIMEVAL, &TIMEZONE);
    double endTime = (TIMEVAL.tv_sec + TIMEVAL.tv_usec*1.0e-6);
    //performance on CPU
	printf("\n\n***********************************************************************\n");
    printf("Time taken on CPU (sec) = %lf \n",endTime- startTime);
    printf("GFLOPS = %.5f\n", (2L * dim * dim * (dim-1))/ (1e9*(endTime- startTime)));
	printf("\n\n***********************************************************************\n");
}
