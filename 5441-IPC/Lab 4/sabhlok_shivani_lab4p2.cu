#include "hip/hip_runtime.h"
/*
This is a cuda parallel vesion program of sobel operator on GPU
Kernel is launched with the read input image and grid to write to for output along with threshold in each iteration.
*/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "time.h"
#include <math.h>
extern "C" {
#include "read_bmp.h"
}
void GPURelatedStuff(uint8_t *bmp_data,uint8_t *new_bmp_img, int size, uint32_t height, uint32_t width);
#define MAX_ALLOWED_CMD_ARGS  3
#define MIN_RGB_VALUE 0
#define MAX_RGB_VALUE 255
//kernel thread and block configuration combinations
int blocks[] = {64,128,64};
int threadsPerBlock[] = {64,128,32};

//kernel code
__global__ void kernel_sobel_filter(const uint8_t * device_input_data, uint8_t * device_output_data, uint32_t threshold, const uint32_t height, const uint32_t width) {
    //width is number of columns
    //height is number of rows
    int tid = threadIdx.x;
    for (int row = tid; row < height - 1; row += blockDim.x) {
        for (int block = blockIdx.x; block < width - 1; block += gridDim.x) {
            int idx = row * width + block;
            //skip 0th and 0th column
            if (idx >= 0 && idx < width)
				continue;
            if (idx % width == 0)
                continue;
			int neighborIndex[][3] = {{idx - width - 1, idx - width, idx - width + 1},
                                      {idx - 1,         idx,         idx + 1},
                                      {idx + width - 1, idx + width, idx + width + 1}};
            //To detect horizontal lines. This is effectively the gx.
            const int sobel_x[3][3] = {{-1, 0, 1},
                                       {-2, 0, 2},
                                       {-1, 0, 1}};
            // To detect vertical lines. This is effectively the gy.
            const int sobel_y[3][3] = {{1,  2,  1},
                                       {0,  0,  0},
                                       {-1, -2, -1}};
            float magnitude_x = 0.0;
            float magnitude_y = 0.0;
			//applying sobel stencil
            for (int i = 0; i < 3; ++i) {
                for (int j = 0; j < 3; ++j) {
                    magnitude_x += device_input_data[neighborIndex[i][j]] * sobel_x[i][j];
                    magnitude_y += device_input_data[neighborIndex[i][j]] * sobel_y[i][j];
                }
            }

            uint32_t mag = __double2int_ru(sqrt((double)(magnitude_x * magnitude_x + magnitude_y * magnitude_y)));
			if(mag > threshold)
            {
                 device_output_data[idx] = MAX_RGB_VALUE;
            }
            else
            {
				device_output_data[idx]= MIN_RGB_VALUE;
            }
            //device_output_data[idx] = mag;
        }
    }
}

int main(int argc, char* argv[])
{
    int cmd_arg;
    uint8_t *bmp_data;
    uint8_t *new_bmp_img;
    uint32_t wd, ht;
    FILE *out_file, *inFile;
    uint32_t total_cells;

    /*Check if no of arguments is permissible to execute*/
    if (argc != MAX_ALLOWED_CMD_ARGS)
    {
        fprintf(stderr, "Missing input file name and/or output file name in command line parameters\n");
        exit(EXIT_FAILURE);
    }
    /*Reading command line arguments and obtain the values*/
    for (cmd_arg = 1; cmd_arg < argc; cmd_arg++)
    {
        switch (cmd_arg)
        {
            case 1:
                inFile = fopen(argv[cmd_arg], "rb");
                break;

            case 2:
                out_file = fopen(argv[cmd_arg], "wb");
                break;
        }
    }
    //Read the binary bmp file into
    bmp_data = (uint8_t *)read_bmp_file(inFile);
    //Allocate space for new sobel image
    new_bmp_img = (uint8_t *)malloc(get_num_pixel());
    wd = get_image_width();
    ht = get_image_height();
	total_cells = wd * ht;
	//do GPU related memory allocation and other initializations
    GPURelatedStuff(bmp_data,new_bmp_img,total_cells,ht,wd);
    //Write the buffer into the bmp file
    write_bmp_file(out_file, new_bmp_img);

    return 0;
}

void GPURelatedStuff(uint8_t *bmp_data,uint8_t *new_bmp_img, int size, uint32_t height, uint32_t width)
{
	//initialize timing constructs
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    uint8_t * device_input_data;
    uint8_t * device_output_data;
    hipMalloc((void **) & device_input_data, size);
    hipMalloc((void **) & device_output_data, size);
    /* Copy the input data to the device. */
    hipMemcpy(device_input_data, bmp_data, size, hipMemcpyHostToDevice);
	printf("\n\n***********************************************************************\n");
    /* Launch the kernel! */
	for(int i = 0; i< 3;i++)
    {
		dim3 dimGrid(blocks[i]);
		dim3 dimBlock(threadsPerBlock[i]);
		double percent_black_cells = 0;
		uint32_t total_cells;
		int threshold   = 0;
		total_cells = width * height;
		hipEventRecord(start, 0);
		while(percent_black_cells < 75) {
			percent_black_cells = 0;
			threshold += 1;
			kernel_sobel_filter<<<dimGrid, dimBlock>>>(device_input_data,device_output_data,threshold, height, width);
			hipDeviceSynchronize();
			hipMemcpy(new_bmp_img, device_output_data, size, hipMemcpyDeviceToHost);
			hipDeviceSynchronize();
			for(uint32_t i=1; i < (height-1); i++) {
				for (uint32_t j = 1; j < (width-1); j++) {
					if (new_bmp_img[i * width + j] == 0) {
						percent_black_cells++;
					}
				}
			}
			percent_black_cells = (percent_black_cells * 100.0) / (double)total_cells;
		}
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		printf ("With %d blocks and %d threads per block, Time for converging with %d threshold on kernel: %f s\n", blocks[i],threadsPerBlock[i],threshold,time/1e3);
	}
	printf("\n\n***********************************************************************\n");
    hipFree(device_input_data);
    hipFree(device_output_data);
}

